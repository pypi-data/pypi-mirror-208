#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include "diamond.h"
#include "field.h"
#include "testutils.h"

namespace field {
namespace {

using defs::IsAux;
using diamond::X;
using diamond::Y;
using diamond::Z;

// Just a simple point test.
TEST(Field, Field) {
  XY domain(10, 10);
  int npml = 5;
  int zshift = 10;
  int nout = 4;

  testutils::Array<int> arr(ExternalElems<int>(domain, nout, npml));
  Cell<int> cell;
  InitCell(cell, 0);
  cell.Set(42, Node(0, 0, 1, diamond::E, diamond::X));
  XY pos(5, 5);
  int threadpos(3);
  int outindex = 2;
  WriteCell(cell, arr.Ptr(), pos, outindex, threadpos, domain, npml, zshift,
            /*isaux=*/false);
  EXPECT_EQ(arr[ExtNodeIndex(Node(5, 5, 51, diamond::E, diamond::X), outindex,
                             /*xx=*/10,
                             /*yy=*/10, ExtZz<int>(npml))],
            42);
}

TEST(Field, FieldHalf2) {
  XY domain(10, 10);
  int nout = 4;
  int npml = 5;
  int zshift = 7;
  testutils::Array<float> arr(ExternalElems<half2>(domain, nout, npml));
  Cell<half2> cell;
  InitCell(cell, defs::Zero<half2>());
  cell.Set(__floats2half2_rn(42.0f, 43.0f),
           Node(0, 0, 1, diamond::E, diamond::X));
  XY pos(5, 5);
  int threadpos(1);
  int outindex = 2;
  WriteCell(cell, arr.Ptr(), pos, outindex, threadpos, domain, npml, zshift,
            /*isaux=*/false);
  EXPECT_EQ(arr[ExtNodeIndex(Node(5, 5, 106, diamond::E, diamond::X), outindex,
                             /*xx=*/10, /*yy=*/10, ExtZz<half2>(npml))],
            42.0f);
  // Note that the lo-hi values of the half2 are on "opposite" sides of the
  // wrap-around.
  EXPECT_EQ(arr[ExtNodeIndex(Node(5, 5, 0, diamond::E, diamond::X), outindex,
                             /*xx=*/10, /*yy=*/10, ExtZz<half2>(npml))],
            43.0f);
}

} // namespace
} // namespace field
