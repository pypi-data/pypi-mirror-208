#include <test_add_kernel.cu>
#include <test_add.hh>
#include <assert.h>
#include <iostream>

void kernel_add_one_host(int * data_host, int N){
    int size = N*sizeof(int);

    int * data_device;

    hipError_t err = hipMalloc((void**) &data_device, size);
    assert(err == 0);
    err = hipMemcpy(data_device, data_host, size, hipMemcpyHostToDevice);
    assert(err == 0);


    kernel_add_one<<<64, 64>>>(data_device, N);
    err = hipGetLastError();
    assert(err == 0);


    err = hipMemcpy(data_host, data_device, size, hipMemcpyDeviceToHost);
    assert(err == 0);

    hipFree(data_device);
}

float kernel_sum_sin_host(float * data_host, int N){
    int size = N*sizeof(float);

    float res_sum = 0;

    float * data_device;
    float * res_sum_device;

    hipError_t err;

    err = hipMalloc((void**) &data_device, size);
    assert(err == 0);
    err = hipMalloc((void**) &res_sum_device, sizeof(float));
    assert(err == 0);
    
    err = hipMemcpy(data_device, data_host, size, hipMemcpyHostToDevice);
    assert(err == 0);
    err = hipMemcpy(res_sum_device, &res_sum, sizeof(float), hipMemcpyHostToDevice);
    assert(err == 0);

    kernel_sum_sin<<<64, 64>>>(data_device, N, res_sum_device);
    err = hipGetLastError();
    assert(err == 0);

    err = hipMemcpy(&res_sum, res_sum_device, sizeof(float), hipMemcpyDeviceToHost);
    assert(err == 0);

    hipFree(res_sum_device);
    hipFree(data_device);

    return res_sum;
}